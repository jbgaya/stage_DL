
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
using namespace std;
class Vector;
struct LazyExp;



//Classe vecteur___________________________________________PAS IMPORTANT
class Vector{
	public:
		int size;
		double * data;
		
		Vector(int n=0);
		~Vector(){delete [] data;}
		Vector(const Vector &);
		
		double operator[](int i) const {return data[i];}
		double & operator[](int i) {return data[i];}
		
		Vector & operator=(const Vector &);
		Vector & operator=(const LazyExp & E);
			
};

Vector::Vector(int n){
	data = new double[n];
	for(int i=0; i<n; i++) data[i] = 0.; 
}

Vector & Vector::operator=(const Vector & A){
	if(this == &A){return *this;}
	else{
		delete [] data;
		data = new double[A.size];
		for(int i=0; i<size; i++) data[i] = A.data[i]; 
	}
	return *this;
}

Vector::Vector(const Vector & A): size(A.size) {
	data = new double[size];
	for(int i=0;i<size;i++) data[i] = A.data[i];
}


//_________________________________________________________METHODE NAÏVE

vector<double> operator+(const vector<double> & a, const vector<double> & b){
	vector<double> c(a.size());
	for(int i=0; i < c.size(); i++) c[i] = a[i] + b[i];
	return c;
}


//_______________________________________________________LAZY EXPRESSION

struct LazyExp{
	const Vector & A;
	const Vector & B;
	
	LazyExp(const Vector & A_, const Vector & B_): A(A_), B(B_) {};
};

//là où se fait l'évaluation
inline Vector & Vector::operator=(const LazyExp & E){
	for(int i=0; i < E.A.size; i++) data[i] = E.A.data[i] + E.B.data[i];
	return *this;
}

//pas d'évaluation
inline LazyExp operator+(const Vector & A, const Vector & B) {
	return LazyExp(A,B);
}

//______________________________________________________________________


int main(void){

	//initialisation
	mt19937 G;
	uniform_real_distribution<double> U(-1.,1.);
	int n = 10000000;
	Vector A(n), B(n), C(n);
	vector<double> a(n), b(n), c(n);
	for(int i = 0; i < n; i++){
		A[i] = U(G);
		B[i] = U(G);
		a[i] = A[i];
		b[i] = B[i];
	}
	cout << "--- Calcul de A + B (2 vecteurs de taille " << n << ") par différentes méthodes ---" << endl;
	
	
	//méthode naïve
	auto t1 = chrono::system_clock::now();
	c = a + b;
	auto t2 = chrono::system_clock::now();
	chrono::duration<double> diff = t2 - t1;
	cout << " \nTemps de calcul méthode naïve :" << diff.count() << endl;
	
	//lazy expression
	t1 = chrono::system_clock::now();
	C = A + B;
	t2 = chrono::system_clock::now();
	diff = t2 - t1;
	cout << " \nTemps de calcul lazy expression :" << diff.count() << endl;
	
}
